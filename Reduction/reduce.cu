#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

inline void CHECK(const hipError_t error)
{
	if(error != hipSuccess)
	{
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);
		fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error));
		exit(1);
	}
}

double cpuTimer()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void initialData(float *arr, int size)
{
	time_t t;
	srand((unsigned)time(&t));  // seed
	for(int i=0;i<size;i++)
		arr[i]=(float)(rand())/RAND_MAX;
}

__global__ void reduceInterleaved(float *g_idata, float *g_odata, unsigned int n)
{
	unsigned int tid=threadIdx.x;
	unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;

	// convert global data pointer to the local pointer of this block
	float *idata = g_idata + blockIdx.x*blockDim.x;

	// boundary check
	if(idx>=n) return;

	// in-place reduction in global memory
	for(int stride=blockDim.x/2;stride>0;stride>>=1)
	{
		if(tid<stride)
			idata[tid]+= idata[tid+stride];
		__syncthreads();
	}
	// write result for this block to global mem
	if(tid==0) g_odata[blockIdx.x]=idata[0];
}

__global__ void reduceNeighbored(float *g_idata, float *g_odata, unsigned int n)
{
	// set thread ID
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;

	// convert global data pointer to the local pointer of this block
	float *idata = g_idata + blockIdx.x*blockDim.x;

	// boundary check
	if(idx>=n) return;

	// in-place reduction in global memory
	for(int stride=1;stride<blockDim.x; stride *=2)
	{
		if((tid%(2*stride))==0)
			idata[tid] += idata[tid+stride];
		// synchrnoize within block
		__syncthreads();
	}
	// write result for this block to global mem
	if(tid==0) g_odata[blockIdx.x] = idata[0];
}

float recursiveReduce(float *data, int const size)
{
	// terminate check
	if(size==1) return data[0];

	// renew the stride
	int const stride = size / 2;

	// in-place reduction
	for(int i=0;i<stride;i++)
		data[i] += data[i+stride];
	// call recursively
	return recursiveReduce(data, stride);
}

int main(void)
{
	bool bResult = false;
	float cpu_sum, gpu_sum;
	
	// initialize
	int size = 1<<24;	// 16M
	printf("array size %d\n", size);

	// execution configuration
	int blocksize = 512;
	dim3 block(blocksize,1);
	dim3 grid((size+block.x-1)/block.x,1);

	// allocate host memory
	size_t bytes = size*sizeof(float);
	float *h_idata = (float *)malloc(bytes);
	float *h_odata = (float *)malloc(grid.x*sizeof(float));

	// allocate device memory
    float *d_idata, *d_odata;
    hipMalloc((float**)&d_idata, bytes);
    hipMalloc((float**)&d_odata, grid.x*sizeof(float));

	initialData(h_idata, size);
	hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);

	// cpu reduction
	double iStart = cpuTimer();
	cpu_sum = recursiveReduce(h_idata, size);
	double ElapsedTime = cpuTimer()-iStart;
	printf("CPU reduction : %f, Elapsed Time : %f sec\n", cpu_sum, ElapsedTime);

	/********** GPU **************/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
//	reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
	reduceInterleaved<<<grid, block>>>(d_idata, d_odata, size);
	hipDeviceSynchronize();
	hipMemcpy(h_odata, d_odata, grid.x*sizeof(float), hipMemcpyDeviceToHost);
	
	for(int i=0;i<grid.x;i++) gpu_sum += h_odata[i];
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ETime;
	hipEventElapsedTime(&ETime, start, stop);
	printf("gpu reduction : %f, Elapsed Time : %f sec\n", gpu_sum, ETime*1e-3f);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;

}


