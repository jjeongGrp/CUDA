
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ void PrintArray(int tid, int *A)
{
    printf("A[%d]=%d\n",tid,A[tid]);
	if(tid==0) printf("======================\n");
}
__global__ void Print(int *A)
{
	int tid=threadIdx.x;
	PrintArray(tid,A);
}
int main()
{
	int *d1_A, *d2_A, *h_A, *h_B;
	int size=5;
	int i;
	h_A=(int*)malloc(size*sizeof(int));
	h_B=(int*)malloc(size*sizeof(int));
	for(i=0;i<size;i++) h_A[i]=i;
	// Allocate Device memories............	
	hipSetDevice(0);
	hipMalloc((int**)&d1_A,size*sizeof(int));
	hipSetDevice(1);
	hipMalloc((int**)&d2_A,size*sizeof(int));
	//.........................................
	
	// Data Transfer : Host -> device 0
	hipSetDevice(0);
	hipMemcpy(d1_A,h_A,size*sizeof(int), hipMemcpyHostToDevice);
//	cudaMemcpy(d1_A,h_A,size*sizeof(int), cudaMemcpyDefault);
	Print<<<1,5>>>(d1_A);
	hipDeviceSynchronize();

	// Data Transfer : Device 0 -> Device 1
	hipMemcpy(d2_A,d1_A,size*sizeof(int), hipMemcpyDeviceToDevice);
//	cudaMemcpy(d2_A,d1_A,size*sizeof(int), cudaMemcpyDefault);
	hipSetDevice(1);
	Print<<<1,5>>>(d2_A);
	hipDeviceSynchronize();

	// Data Transfer : Device 2 -> Host
	hipMemcpy(h_B,d2_A,size*sizeof(int),hipMemcpyDeviceToHost);
//	cudaMemcpy(h_B,d2_A,size*sizeof(int),cudaMemcpyDefault);
 	for(i=0;i<size;i++) printf("h_B[%d]=%d\n",i,h_B[i]);
	hipFree(d1_A); hipFree(d2_A);
	hipDeviceReset();
	return 0;
}
