#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

inline void CHECK(const hipError_t error)
{
	if(error !=hipSuccess)
	{
		fprintf(stderr, "Error: %s:%d, ",__FILE__,__LINE__);
		fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error));
		exit(1);
	}
}
double cpuTimer()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void initialData(float *arr, const int size)
{
	time_t t;
	srand((unsigned)time(&t));
	for(int i=0;i<size;i++)
		arr[i]= (float)(rand())/RAND_MAX;
}
void MatMulOnCPU(float *A, float *B, float *C, const int Arows, const int Acols, const int Bcols)
{
	float sum;
	for(int i=0;i<Arows;i++)
	{
		for(int j=0;j<Bcols;j++)
		{
			sum = 0.0f;
			for(int k=0;k<Acols;k++)
			{
				sum += A[i*Acols+k]*B[k*Bcols+j];
			}
			C[i*Bcols+j]=sum;
		}
	}
}
//__global__ void MatMultOnGPU(float *A, float *B, float *C, const int nrows, const int ncols)
__global__ void MatMultOnGPU(float *A, float *B, float *C, const int Arows, const int Acols, const int Bcols)
{
	int tx = blockDim.x*blockIdx.x + threadIdx.x;	// col of C 
	int ty = blockDim.y*blockIdx.y + threadIdx.y;	// row of C 
	int tid = ty*Bcols+tx;


	float sum=0.0f;
	if(tx < Bcols && ty <Arows )
	{
		for(int i=0;i<Acols;i++)
		{
			sum += A[ty*Acols + i]*B[i*Bcols+tx];
		}
		C[tid]=sum;
	}
}

void checkResult(float *host, float *gpu, const int N)
{
	double epsilon = 1.0e-8;
	bool match = 1;
	for(int i=0;i<N;i++)
	{
		if(abs(host[i]-gpu[i])>epsilon)
		{
			match = 0;
			printf("Matrices do not match!\n");
			printf("host %10.7f, gpu %10.7f at current %d\n", host[i], gpu[i], i);
			break;
		}
	}
	if(match)printf("Matrices match.\n");
}
int main(int argc, char **argv)
{
	double Start, ElapsedTime;
	float *MatA, *MatB, *MatC, *gpu_MatC;
	int Arows=300, Acols=200, Bcols=400;
	int threads_x=32, threads_y=32;
	if(argc>1) Arows=atoi(argv[1]);
	if(argc>2) Acols=atoi(argv[2]);
	if(argc>3) Bcols=atoi(argv[3]);
	if(argc>4) threads_x = atoi(argv[4]);
	if(argc>5) threads_y = atoi(argv[5]);
	/************ ON CPU **************/
	MatA=(float*)malloc(Arows*Acols*sizeof(float));
	MatB=(float*)malloc(Acols*Bcols*sizeof(float));
	MatC=(float*)malloc(Arows*Bcols*sizeof(float));
	gpu_MatC=(float*)malloc(Arows*Bcols*sizeof(float));
	
	initialData(MatA, Arows*Acols);
	initialData(MatB, Acols*Bcols);
	
	Start=cpuTimer();
	MatMulOnCPU(MatA, MatB, MatC, Arows, Acols, Bcols);
	ElapsedTime=cpuTimer()-Start;
	printf("Elapsed Time on CPU : %f\n",ElapsedTime);
	/**********************************/

	/************ ON GPU **************/
	float *d_MatA, *d_MatB, *d_MatC;
	CHECK(hipMalloc((float**)&d_MatA, Arows*Acols*sizeof(float)));
	CHECK(hipMalloc((float**)&d_MatB, Acols*Bcols*sizeof(float)));
	CHECK(hipMalloc((float**)&d_MatC, Arows*Bcols*sizeof(float)));

	Start=cpuTimer();
	CHECK(hipMemcpy(d_MatA,MatA, Arows*Acols*sizeof(float),hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_MatB,MatB, Acols*Bcols*sizeof(float),hipMemcpyHostToDevice));
	dim3 block(threads_x,threads_y,1);
	dim3 grid((Bcols+block.x-1)/block.x, (Arows+block.y-1)/block.y, 1);
	MatMultOnGPU<<<grid, block>>>(d_MatA, d_MatB, d_MatC, Arows, Acols, Bcols);

	CHECK(hipMemcpy(gpu_MatC, d_MatC, Arows*Bcols*sizeof(float), hipMemcpyDeviceToHost));
	ElapsedTime=cpuTimer()-Start;
	printf("Elapsed Time on GPU : %f\n",ElapsedTime);
	/**********************************/
	checkResult(MatC, gpu_MatC, Arows*Bcols);

	free(MatA),	free(MatB),	free(MatC),	free(gpu_MatC);
	CHECK(hipFree(d_MatA)), CHECK(hipFree(d_MatB)), CHECK(hipFree(d_MatC));

	CHECK(hipDeviceReset());
	return 0;
}
